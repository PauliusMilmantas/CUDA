#include "hip/hip_runtime.h"
#include <stdio.h>

#define BLOCK_SIZE 256

__global__ void parallelSum(int* inputArray, int* output) {
    // Declare a shared memory array
    __shared__ int partialSum[BLOCK_SIZE];

    // Calculate global thread ID
    int tid = threadIdx.x + blockDim.x *  blockIdx.x;

    // Each thread loads an element from global memory to shared memory
    partialSum[threadIdx.x] = inputArray[tid];

    // Synchronize to make sure all threads have finished copying
    __syncthreads();

    // Perform parallel reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            partialSum[threadIdx.x] += partialSum[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // The first thread in the block writes the result to global memory
    if (threadIdx.x == 0) {
        output[blockIdx.x] = partialSum[0];
    }
}

int main() {
    // Set the size of the array
    int arraySize = 1024;

    // Allocate and initialize the array on the host
    int* h_inputArray = (int*)malloc(arraySize * sizeof(int));
    for (int i = 0; i < arraySize; ++i) {
        h_inputArray[i] = i;
    }

    // Allocate memory on the device
    int* d_inputArray, * d_output;
    hipMalloc((void**)&d_inputArray, arraySize * sizeof(int));
    hipMalloc((void**)&d_output, sizeof(int));

    // Copy the input array from the host to the device
    hipMemcpy(d_inputArray, h_inputArray, arraySize * sizeof(int), hipMemcpyHostToDevice);

    // Calculate grid and block sizes
    dim3 blockSize(BLOCK_SIZE, 1, 1);
    dim3 gridSize((arraySize + blockSize.x - 1) / blockSize.x, 1, 1);

    // Launch the kernel
    parallelSum << <gridSize, blockSize >> > (d_inputArray, d_output);

    // Copy the result back to the host
    int h_output;
    hipMemcpy(&h_output, d_output, sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    printf("Sum: %d\n", h_output);

    // Free allocated memory
    free(h_inputArray);
    hipFree(d_inputArray);
    hipFree(d_output);

    return 0;
}